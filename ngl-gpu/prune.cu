#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>

#include <ANN/ANN.h>

__global__
void prune_discrete(const int N, const int D, const int K, const int steps,
                    float *X, int *edgesIn, int *edgesOut, float *erTemplate)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_x = blockDim.x * gridDim.x;

    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_y = blockDim.y * gridDim.y;
    
    // References to points in X
    float *p, *q, *r;

    //TODO: Fix this
    // Computed vectors representing the edge under test pq and the vector from
    // one end point to a third point r (We will iterate over all possible r's)
    float pq[2] = {};
    float pr[2] = {};

    // Different iterator/indexing variables i, j, and n are rows in X
    // representing p, q, and r, respectively
    // k is the nearest neighbor, d is the dimension
    int i, j, k, k2, d, n;

    // t is the parameterization of the projection of pr onto pq
    // In layman's terms, this is the length of the shadow pr casts onto pq
    // lookup is the 
    float t;
    int lookup;

    // Some other temporary variables
    float length_squared;
    float squared_distance_to_edge;
    float minimum_allowable_distance;

    for (k = index_y; k < K; k += stride_y) {
        for (i = index_x; i < N; i += stride_x) {

            p = &(X[D*i]);
            j = edgesIn[K*i+k];
            q = &(X[D*j]);
        
            length_squared = 0;
            for(d = 0; d < D; d++){
                pq[d] = p[d] - q[d];
                length_squared += pq[d]*pq[d];
            }
            // A point should not be connected to itself
            if(length_squared == 0) {
                edgesOut[K*i+k] = -1;
                continue;
            }
            
            // for(n = 0; n < N; n++) {
            for(k2 = 0; k2 < 2*K; k2++) {
                n = (k2 < K) ? edgesIn[K*i+k2] : edgesIn[K*j+(k2-K)];
                r = &(X[D*n]);

                t = 0;
                for(d = 0; d < D; d++){
                    pr[d] = p[d] - r[d];
                    t += pr[d]*pq[d];
                }

                t /= length_squared;
                lookup = __float2int_rd(abs(steps * (2 * t - 1))+0.5);
                if (lookup >= 0 && lookup <= steps) {
                    squared_distance_to_edge = 0;
                    for(d = 0; d < D; d++){
                        squared_distance_to_edge += (pr[d] - pq[d]*t)*(pr[d] - pq[d]*t);
                    }    
                    minimum_allowable_distance = sqrt(length_squared)*erTemplate[lookup];

                    if(sqrt(squared_distance_to_edge) < minimum_allowable_distance) {
                        edgesOut[K*i+k] = -1;
                        break;
                    }
                }
            }
        }
    }
}

__global__
void prune(const int N, const int D, const int K, const int steps, float *X,
           int *edgesIn, int *edgesOut, float lp, float beta)
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_x = blockDim.x * gridDim.x;

    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_y = blockDim.y * gridDim.y;
    
    float *p, *q, *r;

    float pq[2] = {};
    float pr[2] = {};

    int i, j, k, k2, d, n;
    float t;

    float length_squared;
    float squared_distance_to_edge;
    float minimum_allowable_distance;

    ////////////////////////////////////////////////////////////
    float xC, yC, radius, y;
    ////////////////////////////////////////////////////////////

    for (k = index_y; k < K; k += stride_y) {
        for (i = index_x; i < N; i += stride_x) {
            p = &(X[D*i]);
            j = edgesIn[K*i+k];
            q = &(X[D*j]);
        
            length_squared = 0;
            for(d = 0; d < D; d++){
                pq[d] = p[d] - q[d];
                length_squared += pq[d]*pq[d];
            }
            // A point should not be connected to itself
            if(length_squared == 0) {
                edgesOut[K*i+k] = -1;
                continue;
            }
            
            // for(n = 0; n < N; n++) {
            for(k2 = 0; k2 < 2*K; k2++) {
                n = (k2 < K) ? edgesIn[K*i+k2] : edgesIn[K*j+(k2-K)];
                r = &(X[D*n]);

                // t is the parameterization of the projection of pr onto pq
                // In layman's terms, this is the length of the shadow pr casts onto pq
                t = 0;
                for(d = 0; d < D; d++){
                    pr[d] = p[d] - r[d];
                    t += pr[d]*pq[d];
                }

                t /= length_squared;

                if (t > 0 && t < 1) {
                    squared_distance_to_edge = 0;
                    for(d = 0; d < D; d++){
                        squared_distance_to_edge += (pr[d] - pq[d]*t)*(pr[d] - pq[d]*t);
                    }

                    ////////////////////////////////////////////////////////////
                    // ported from python function, can possibly be improved 
                    // in terms of performance
                    xC = 0;
                    yC = 0;

                    if (beta <= 1) {
                        radius = 1. / beta;
                        yC = powf(powf(radius, lp) - 1, 1. / lp);
                    }
                    else {
                        radius = beta;
                        xC = 1. - beta;
                    }
                    t = fabs(2*t-1);
                    y = powf(powf(radius, lp) - powf(t-xC, lp), 1. / lp) - yC;
                    minimum_allowable_distance = 0.5*y*sqrt(length_squared);

                    //////////////////////////////////////////////////////////
                    if(sqrt(squared_distance_to_edge) < minimum_allowable_distance) {
                        edgesOut[K*i+k] = -1;
                        break;
                    }
                }
            }
        }
    }
}

float minDistanceFromEdge(float t, float beta, float p) {
    float xC = 0;
    float yC = 0;
    float r;

    if (t > 1) {
        return 0;
    }
    if (beta <= 1) {
        r = 1. / beta;
        yC = powf(powf(r, p) - 1, 1. / p);
    }
    else {
        r = beta;
        xC = 1 - beta;
    }
    float y = powf(powf(r, p) - powf(t-xC, p), 1. / p) - yC;
    return 0.5*y;
}

void createTemplate(float * data, float beta=1, int p=2, int steps=100) {
    if (p < 0) {
        if (beta >= 1) {
            for (int i = 0; i <= steps; i++) {
                data[i] = beta / 2.;
            }
        }
        else {
            for (int i = 0; i <= steps; i++) {
                data[i] = 0.;
            }
        }
    }
    else {
        for (int i = 0; i <= steps; i++) {
            data[i] = minDistanceFromEdge(float(i)/steps, beta, p);
        }
    }
}


int main(int argc, char **argv)
{
  struct hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, 0);
  std::cout << "using " << properties.multiProcessorCount << " multiprocessors"
            << std::endl 
            << "max threads per processor: " 
            << properties.maxThreadsPerMultiProcessor << std::endl;

  std::string pointFile = "../data_2_1000000_0.csv";
//   std::string edgeFile = "../knn_2D_1000000.txt";

  int N = 1000000;
  int D = 2;
  int K = 100;
  int steps = 9999;
  bool discrete = false;

  // Load data set and edges from files
  // TODO
  float *x;
  int *edgesIn;
  int *edgesOut;
  float *referenceShape;
  dim3 blockSize(32, 32);
  dim3 gridSize(4, 4);

  int i, d, k;

  std::string outputFilename;

  ANNpointArray dataPts;
  ANNpoint queryPt;
  ANNidxArray nnIdx;
  ANNdistArray dists;
  ANNkd_tree* kdTree;

  dataPts = annAllocPts(N, D);

  hipMallocManaged(&x, N*D*sizeof(float));
  hipMallocManaged(&edgesIn, N*K*sizeof(int));
  hipMallocManaged(&edgesOut, N*K*sizeof(int));
  hipMallocManaged(&referenceShape, (steps+1)*sizeof(float));

  std::ifstream file1( pointFile );
  
  i = 0;
  d = 0;
  std::string line;
  
  while ( std::getline(file1, line) )
  {
    std::istringstream iss(line);
    for (d = 0; d < D; d++) {
      iss >> x[i*D+d];
      dataPts[i][d] = x[i*D+d];
    }
    i++;
  }
  file1.close();
  
  kdTree = new ANNkd_tree(dataPts, N, D);
  queryPt = annAllocPt(D);
  nnIdx = new ANNidx[K];
  dists = new ANNdist[K];
  for(i = 0; i < N; i++) {
    for(d = 0; d < D; d++) {
        queryPt[d] = x[i*D+d];
    }
    kdTree->annkSearch(queryPt, K, nnIdx, dists, 0.f);
    for(int k=0;k<K;k++) {
        edgesOut[i*K+k] = edgesIn[i*K+k] = nnIdx[k];
    }
  }

  annDeallocPts(dataPts);
  annDeallocPt(queryPt);
  delete nnIdx;
  delete dists;
  delete kdTree;

//   std::ifstream file2 ( edgeFile );
//   i = 0;
//   k = 0;
//   while ( std::getline(file2, line) )
//   {
//     std::istringstream iss(line);
//     for (k = 0; k < K; k++) {
//       iss >> edgesIn[i*K+k];
//       edgesOut[i*K+k] = edgesIn[i*K+k];
//     }
//     i++;
//   }
//   file2.close();

  if(discrete) {
    createTemplate(referenceShape, 1, 2, steps);
    prune_discrete<<<gridSize, blockSize>>>(N, D, K, steps, x, edgesIn, edgesOut, referenceShape);
    outputFilename = "/home/maljovec/projects/active/ngl/edges_2D_gpu_discrete.txt";
  }
  else {
    prune<<<gridSize, blockSize>>>(N, D, K, steps, x, edgesIn, edgesOut, 2, 1);
    outputFilename = "/home/maljovec/projects/active/ngl/edges_2D_gpu.txt";
  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));

  hipDeviceSynchronize();
  
  std::ofstream file5 (outputFilename);
  for(i = 0; i < N; i++) {
    for(k = 0; k < K; k++) {
      if (edgesOut[i*K+k] != -1) {
        file5 << i << " " << edgesOut[i*K+k] << std::endl;
      }
    }
  }

  // Free memory
  hipFree(x);
  hipFree(edgesIn);
  hipFree(edgesOut);

  return 0;
}